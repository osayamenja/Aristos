/******************************************************************************
 * Copyright (c) 2024, Osayamen Jonathan Aimuyo.
 ******************************************************************************/
#include <fmt/ranges.h>
#include <thrust/generate.h>
#include <thrust/random.h>

#include "correctness.cuh"
#include "include/bootstrap.cuh"
#include "include/moe/moe.cuh"

__host__ __forceinline__
void runOS() {
    aristos::initialize();
    const auto rank = aristos::getRank();
    // generate random input tile and eye weights
    constexpr auto S = aristos::ACC::S::value;
    constexpr auto H = aristos::ACC::H::value;
    constexpr auto E = aristos::ACC::E::value;
    constexpr auto P = aristos::ACC::P::value;
    constexpr auto PX = aristos::ACC::PX::value;
    const auto nLx = aristos::hostBookkeeping.nLx;
    constexpr unsigned long aZ =  S * H;
    constexpr auto gwZ = aZ + PX * H;
    // scale this to number of experts
    const auto bZ =  gwZ + nLx * P * H;
    const auto b2Z =  bZ + nLx * P * H;
    const auto dZ =  b2Z + nLx * (P + H);
    const auto gZ = dZ + S * PX;
    const auto cZ = gZ + S * H;
    cuda::std::byte* p;
    ARISTOS_CHECK_CUDA(hipMallocAsync(&p, cZ * sizeof(float), aristos::aristosStream));
    ARISTOS_CHECK_CUDA(hipMemsetAsync(p, 0, cZ * sizeof(float), aristos::aristosStream));
    auto* hP = std::calloc(cZ, sizeof(float));
    auto* fHp = static_cast<float*>(hP);
    using Element = aristos::ACC::Element;
    auto* __restrict__ eHp = static_cast<Element*>(hP);
    {
        #if ARISTOS_NVTX
        aristos::aristosRange forwardRange{"Host Data Prep"};
        #endif
        thrust::default_random_engine rng(47 * (rank + 42));
        thrust::normal_distribution<float> dist(0, 5);
        // Activations
        thrust::generate(fHp, fHp + aZ, [&] { return dist(rng); });
        // gate weights
        thrust::generate(fHp + aZ, fHp + aZ + E * H, [&] { return dist(rng); });
        // Expert weights
        // loop for number of experts
        for (uint i = 0; i < nLx; ++i) {
            // expert up
            thrust::generate(fHp + gwZ + i * (P * H), fHp + gwZ + (i + 1) * (P * H),
                [&] { return dist(rng); });
            thrust::generate(fHp + bZ + i * (P * H), fHp + bZ + (i + 1) * (P * H),
                [&] { return dist(rng); });
        }
        // bias
        std::ranges::fill(fHp + b2Z, fHp + dZ, 0.0f);
        constexpr cutlass::NumericConverter<Element, float> conv{};
        for (uint i = 0; i < dZ; ++i) {
            eHp[i] = conv(fHp[i]);
        }
    }
    ARISTOS_CHECK_CUDA(hipMemcpyAsync(p, eHp, sizeof(Element) * dZ,
        hipMemcpyHostToDevice,
        aristos::aristosStream));
    float timed = 0;
    aristos::moe::forwardHostBench<32, 32>(p, p + dZ * sizeof(Element), timed);
    printf("epRank: %u took %.2fms\n", aristos::hostBookkeeping.rank, timed);
    ARISTOS_CHECK_CUDA(hipPeekAtLastError());
    aristos::finalize();
    std::free(hP);
}

__host__ __forceinline__
void runReference() {
    constexpr auto S = 32;
    constexpr auto H = 32;
    constexpr auto E = 16;
    constexpr auto P = 32;
    constexpr auto PX = E;
    constexpr unsigned long aZ =  S * H;
    constexpr auto gwZ = aZ + PX * H;
    // scale this to number of experts
    constexpr auto nLx = E;
    constexpr auto bZ =  gwZ + nLx * P * H;
    constexpr auto b2Z =  bZ + nLx * P * H;
    constexpr auto dZ =  b2Z + nLx * (P + H);
    constexpr auto gZ = dZ + S * PX;
    constexpr auto cZ = gZ + S * H;
    void* p;
    ARISTOS_CHECK_CUDA(hipMallocAsync(&p, cZ * sizeof(float), aristos::aristosStream));
    ARISTOS_CHECK_CUDA(hipMemsetAsync(p, 0, cZ * sizeof(float), aristos::aristosStream));
    auto* hP = std::calloc(cZ, sizeof(float));
    auto* fHp = static_cast<float*>(hP);
    using ET = float;
    auto* __restrict__ eHp = static_cast<ET*>(hP);
    auto* __restrict__ dP = static_cast<ET*>(p);
    {
        #if ARISTOS_NVTX
        aristos::aristosRange forwardRange{"Host Data Prep"};
        #endif
        thrust::default_random_engine rng(47 * 42);
        thrust::normal_distribution<float> dist(0, 5);
        // Activations, Gate weights, expert weights
        thrust::generate(fHp, fHp + b2Z, [&] { return dist(rng); });
        if constexpr (!cuda::std::is_same_v<ET, float>) {
            constexpr cutlass::NumericConverter<ET, float> conv{};
            for (uint i = 0; i < dZ; ++i) {
                eHp[i] = conv(fHp[i]);
            }
        }
    }
    ARISTOS_CHECK_CUDA(hipMemcpyAsync(p, eHp, sizeof(ET) * dZ,
        hipMemcpyHostToDevice,
        aristos::aristosStream));
    auto* __restrict__ act = dP;
    auto* __restrict__ gateWeights = dP + aZ;
    auto* __restrict__ expertWeights = dP + gwZ;
    auto* __restrict__ bias = dP + b2Z;
    auto* __restrict__ gateOutput = dP + dZ;
    auto* __restrict__ moeOutput = dP + gZ;
    aristos::rExpert<S, H, P, E>(act,
        gateWeights, expertWeights, bias, gateOutput, moeOutput, nLx);
    ARISTOS_CHECK_CUDA(hipMemcpyAsync(eHp, gateOutput, sizeof(ET) * S * PX, hipMemcpyDeviceToHost,
        aristos::aristosStream));
    ARISTOS_CHECK_CUDA(hipStreamSynchronize(aristos::aristosStream));
    const auto cGo = make_tensor(eHp,
            cute::Layout<cute::Shape<cute::Int<S>, cute::Int<E>>,
            cute::Stride<cute::Int<E>, cute::_1>>{});
    print_tensor(cGo);
}
int main() {
    runOS();
}
