/******************************************************************************
 * Copyright (c) 2024, Osayamen Jonathan Aimuyo.
 ******************************************************************************/
#include <torch/torch.h>

#include "include/moe/moe.cuh"
#include "include/throughput.cuh"
#include "include/types.cuh"

__host__ __forceinline__
void evalExpert() {
    using GPUType = aristos::Hardware<ARISTOS_ARCH, 255>;
    constexpr auto blocks = GPUType::OS::processorBlocks::value;
    constexpr auto M = 8192UL;
    constexpr auto N = 4096UL;
    constexpr auto K = 1024UL;
    static_assert(M % BLOCK_M == 0 && M < BLOCK_M * blocks * 128 &&
        N % BLOCK_N == 0 && K % BLOCK_N == 0);
    using clk = std::chrono::high_resolution_clock;
    std::chrono::duration<float> end {};
    // create torch tensors
    CHECK_ERROR_EXIT(hipSetDevice(0));
    constexpr auto sT = torch::kFloat16;
    at::globalContext().setAllowTF32CuBLAS(true);
    at::globalContext().setAllowTF32CuDNN(true);
    const auto options = torch::TensorOptions().dtype(sT).layout(torch::kStrided).device(torch::kCUDA, 0);
    const torch::Device device(torch::kCUDA, 0);
    torch::nn::Sequential expert(
        torch::nn::Linear(torch::nn::LinearOptions(K, N).bias(true)),
        torch::nn::ReLU(),
        torch::nn::Linear(torch::nn::LinearOptions(N, K).bias(true))
        );
    // set expert weights to identity and bias to zero to avoid accumulation errors for this test
    expert->to(device, sT);

    using ElementAccum = float;
    using Activation = cutlass::epilogue::thread::ReLU<ElementAccum>;
    using Element = cute::half_t;
    constexpr auto aZ =  M * K;
    constexpr auto bZ =  aZ + N * K;
    constexpr auto b2Z =  bZ + N * K;
    constexpr auto dZ =  b2Z + N;
    constexpr auto d2Z =  dZ + K;
    constexpr auto sZ =  d2Z + M;
    constexpr auto cWz =  sZ + M;
    constexpr auto cZ =  cWz + M * N;
    constexpr auto hZ =  cZ + M * K;

    // Pack A, B, D, S into a single, linear tensor
    const auto hT = torch::ones({1, hZ}, options).contiguous();
    const auto activations = torch::ones({M, K}, options);
    const auto scaleWeights = torch::ones({M, 1}, options);
    // Pack A
    hT.index({0, torch::indexing::Slice(torch::indexing::None, aZ)}) =
        activations.view({aZ}).contiguous();
    // Pack expert weights
    hT.index({0, torch::indexing::Slice(aZ, bZ)}) =
        expert->named_parameters()[0].value().view({N * K}).contiguous();
    hT.index({0, torch::indexing::Slice(bZ, b2Z)}) =
        expert->named_parameters()[2].value().view({N * K}).contiguous();
    // Pack expert bias
    hT.index({0, torch::indexing::Slice(b2Z, dZ)}) =
        expert->named_parameters()[1].value().view({N}).contiguous();
    hT.index({0, torch::indexing::Slice(dZ, d2Z)}) =
        expert->named_parameters()[3].value().view({K}).contiguous();
    // Pack Scale
    hT.index({0, torch::indexing::Slice(d2Z, sZ)}) =
        scaleWeights.view({M}).contiguous();
    // set output to zero
    hT.index({0, torch::indexing::Slice(cZ, hZ)}) = torch::zeros({M * K}, options).contiguous();
    const auto combineWeights = hT.index({0, torch::indexing::Slice(sZ, cWz)}).view({M, 1});
    // gemm 1 -> ReLU -> gemm 2 -> scale
    constexpr auto trials = 128U;
    const auto start = clk::now();
    const auto result = mul(expert->forward(activations), scaleWeights);
    CHECK_ERROR_EXIT(hipDeviceSynchronize());
    end = clk::now() - start;
    printf("Torch takes %fms\n", end.count() * 1000);

    // Get a copy of the reference result
    aristos::WorkerAttribute wA{};
    // compute & measure fused expert
    aristos::mFT<GPUType, trials, aristos::CombineMode::single, Activation, aristos::UseBarrier::no>(&wA, M, N, K,
        CAST_TO(Element, hT.mutable_data_ptr()),
        CAST_TO(Element, hT.mutable_data_ptr()) + cWz);
    // verify and compare
    std::cout << "Passed? " << (result.view({M * K})
        .allclose(hT.index({0, torch::indexing::Slice(cZ, hZ)}),
            1e-03, 1e-05, true) ? "Yes!" : "No")
    << std::endl;
    std::cout << result.index({1024,
        torch::indexing::Slice(256, 266)}).view({1, 10})
    << std::endl;
    std::cout << hT.index({0, torch::indexing::Slice(cZ, hZ)}).view({M, K}).
        index({1024, torch::indexing::Slice(256, 266)}).view({1, 10})
    << std::endl;
    CHECK_LAST();
}
/*__host__ __forceinline__
void startAristos() {
    using Element = cute::tfloat32_t;
    // GPT3-MoE 1.3B
    const auto iC = aristos::InitialConfig{
        50 * 1024,
        24,
        512,
        4,
        2,
        2048,
        2048,
        2048 * 4,
        1,
        1,
        16,
        true,
        aristos::ActivationFunction::ReLu,
        true,
    };
    aristos::initialize(iC, torch::kFloat32);
    //aristos::moe::forwardHost();
    aristos::finalize();
}*/
int main() {
    cutlass::AlignedArray<float, 2> f{};
    auto d = static_cast<uint>(f[0]);
    evalExpert();
    return 0;
}
