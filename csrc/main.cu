#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, Osayamen Jonathan Aimuyo.
 ******************************************************************************/
#include <iostream>
#include <array>
#include <atomic>

#include <hip/hip_runtime.h>
#include <cuda/std/array>
#include <cuda/std/chrono>
#include <cuda/atomic>
#include <cuda/barrier>

#include <nvshmemx.h>
#include <nvshmem.h>
#include <host/nvshmemx_api.h>

#include "include/aristos.cuh"
#include <functional>
#include <queue>

#include <hipcub/hipcub.hpp>
#include <cute/tensor.hpp>
#include <cute/config.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/collective/collective_mma.hpp>
#include <boost/pending/disjoint_sets.hpp>

#define THREADS_PER_WARP 32
#define THREADS_PER_BLOCK 256

__device__ __constant__ cuda::atomic<unsigned int, cuda::thread_scope_device> last{1};

#define SEQ 23U
__global__ void benchAtomics(CUTE_GRID_CONSTANT const int iter, unsigned int* flag, bool skip = false, bool shouldPersist = false){
    // initialization
    using Nano = cuda::std::chrono::duration<double, cuda::std::nano>;
    unsigned int* pUnderTest = flag;
    __shared__ unsigned int sharedFlag;
    if(shouldPersist){
        cuda::associate_access_property(flag, cuda::access_property::persisting{});
        //pUnderTest = &sharedFlag;
    }
    __syncthreads();
    atomicExch(&sharedFlag, 0);
    Nano a_flag = Nano::zero(), a_cas = Nano::zero(), a_or = Nano::zero(), a_and = Nano::zero();
    cuda::atomic_ref<unsigned int, cuda::thread_scope_device> aFlag (*pUnderTest);
    CUTE_UNROLL
    for(int i = 0; i < iter; ++i){
        uint64_t start, end;
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        aFlag.load();
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        a_flag += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);

        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        atomicCAS(pUnderTest, 0, 0);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        a_cas += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);

        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        atomicOr(pUnderTest, 0U);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        a_or += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);

        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        atomicAnd(pUnderTest, 1U);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        a_and += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);

    }
    using BlockReduce = hipcub::BlockReduce<Nano, THREADS_PER_BLOCK>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    a_flag = BlockReduce(temp_storage).Reduce(a_flag, hipcub::Max());
    a_cas = BlockReduce(temp_storage).Reduce(a_cas, hipcub::Max());
    a_or = BlockReduce(temp_storage).Reduce(a_or, hipcub::Max());
    a_and = BlockReduce(temp_storage).Reduce(a_and, hipcub::Max());

    if(aristos::block::threadID() == 0 && !skip){
        printf("Block Id is %u, a_flag: {T: %f, V: %d}, a_cas: {T: %f, V:%u}, a_or: {T: %f, V:%u}, a_and: {T: %f, V: %u},"
               "isShared: %s\n",
               aristos::grid::blockID(),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(a_flag / (iter*1.0)).count(),
               aFlag.load(),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(a_cas/(iter*1.0)).count(),
               atomicCAS(pUnderTest, 0, 0),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(a_or / (iter * 1.0)).count(),
               atomicOr(pUnderTest, 0U),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(a_and/(iter*1.0)).count(),
               atomicAnd(pUnderTest, 1U),
               (shouldPersist)? "Yes" : "No");
    }
}

template<unsigned int bM=128, unsigned int bN=128, unsigned int bK=8, unsigned int bP=3>
__global__ void occupancyTestKernel(){
    __shared__ float sharedA[cute::cosize_v<decltype(cute::make_layout(cute::make_shape(cute::Int<bM>{}, cute::Int<bK>{}, cute::Int<bP>{})))>];
    __shared__ float sharedB[cute::cosize_v<decltype(cute::make_layout(cute::make_shape(cute::Int<bN>{}, cute::Int<bK>{}, cute::Int<bP>{})))>];
}

namespace aristos{
    bool isInitialized = false;
    hipStream_t aristosStream = hipStreamPerThread;
    void aristosInit(const unsigned int& seqLen, const unsigned int& embedDim, const unsigned int& hiddenProjDim,
                     const unsigned int& k, const unsigned int& capacityFactor,
                     const unsigned int& numExperts) {
        assert(!isInitialized);
        isInitialized = true;
        int numSMs = 0;
        int numBlocksPerSM = 0;
        constexpr int minCommunicatorBlocks = 2;
        int localRank = 0;
        int computeCapability = 0;
        CUTE_CHECK_ERROR(hipGetDevice(&localRank));
        CUTE_CHECK_ERROR(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, localRank));
        const auto GEMMBlocks = cute::ceil_div(seqLen, bM) * cute::ceil_div(hiddenProjDim, bN);
        const auto minBlocks = GEMMBlocks + minCommunicatorBlocks;
        CUTE_CHECK_ERROR(hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &numBlocksPerSM,
                occupancyTestKernel<bM, bN, bK, bP>,
                blockSize,
                sizeof(aristos::maxPrecision) * ((bK * bP) * (bM + bN))));
        const int maxActiveBlocks = numBlocksPerSM * numSMs;
        assert(minBlocks <= maxActiveBlocks);
        int deviceSupportsMemoryPools = 0;
        CUTE_CHECK_ERROR(hipDeviceGetAttribute(&deviceSupportsMemoryPools,
                                                hipDeviceAttributeMemoryPoolsSupported, localRank));
        assert(deviceSupportsMemoryPools);
        CUTE_CHECK_ERROR(hipDeviceGetAttribute(&computeCapability, hipDeviceAttributeComputeCapabilityMajor, localRank));
        /// Due to NVSHMEM: https://docs.nvidia.com/nvshmem/release-notes-install-guide/install-guide/abstract.html#hardware-requirements
        assert(computeCapability >= 7);

        // Good to go! Let's do some initialization
        // initialize NVSHMEM
        nvshmem_init();
        CUTE_CHECK_ERROR(hipSetDevice(localRank));
        // Run Lysi
        // ...
        // generates the below
        unsigned int numNeighbors = 0;
        unsigned int numLocalExperts = 0;
        std::vector<specType> parallelSpec{};
        std::vector<specType> translation{};

        // Allocate Symmetric Heap + Flags
        auto trailer = k + 2U;
        size_t payload = Config::getCapacity(seqLen, numExperts, capacityFactor, k)
                         * (embedDim + trailer);
        size_t heapBytes = numNeighbors * stages * numCells * payload;
        heapBytes += numNeighbors * (sizeof(flagsType) / sizeof(maxPrecision));
        auto sHeap = nvshmem_calloc(heapBytes, sizeof(maxPrecision));

        // Final Initialization
        hostMoEConfig = Config();
        unsigned int* bookKeeping;
        /// pubQueueLen -> Multiplied by 2 to simulate pair: {index, numTokens}
        /// + translationLen + shardSpecLen +
        /// syncVectorLen -> {syncGrid, checkpoints}
        hostMoEConfig.bookKeepingLen = (numLocalExperts * numNeighbors * 2) + numNeighbors + numExperts + (numNeighbors * 2);
        CUTE_CHECK_ERROR(hipMallocAsync(&bookKeeping,
                                    sizeof(specType)*hostMoEConfig.bookKeepingLen,
                                    aristosStream));
        //TODO init with host memcpy?
        hostMoEConfig.numPublisherBlocks = maxActiveBlocks - GEMMBlocks;
        hostMoEConfig.worldSize = numNeighbors;
        hostMoEConfig.bookKeeping = bookKeeping;
        hostMoEConfig.sHeap = static_cast<cuda::std::byte*>(sHeap);
        CUTE_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(moeConfig),
                                            &hostMoEConfig, sizeof(Config), 0, hipMemcpyHostToDevice, aristosStream));
        CUTE_CHECK_ERROR(hipPeekAtLastError());
        CUTE_CHECK_ERROR(hipStreamSynchronize(aristosStream));
    }

    void forwardHost(){

    }

    void backwardHost(){
    }

    void aristosFinalize(){
        assert(isInitialized);
        isInitialized = false;
        CUTE_CHECK_ERROR(hipFreeAsync(hostMoEConfig.bookKeeping, aristosStream));
        nvshmem_free(hostMoEConfig.sHeap);
        nvshmem_finalize();
        CUTE_CHECK_ERROR(hipPeekAtLastError());
        CUTE_CHECK_ERROR(hipStreamSynchronize(aristosStream));
    }
}

extern constexpr int peers = 1;
extern constexpr int stages = 2;
extern constexpr int cells = 2;

extern constexpr int capacity = 1;
extern constexpr int k = 0;
extern constexpr int embedDim = 0;
extern constexpr int tokens = capacity * (embedDim + k + 1);

extern constexpr int peerStride = stages * cells * tokens;
extern constexpr int stageStride = cells * tokens;
extern constexpr int cellStride = tokens;
extern constexpr int tokenStride = (embedDim + k + 1);
extern constexpr int finalTokenStride = (embedDim + 1);

template<typename T>
CUTE_DEVICE
T* getTokenPointer(T* const& addr, unsigned int const& peer, unsigned int const& stage, unsigned int const& cell, unsigned int const& token){
    return addr + ((peer * peerStride) + (stage * stageStride) + (cell * cellStride) + (token * tokenStride));
}

__global__ void benchTen(unsigned int* foo, bool skip = false, bool shouldPersist = false){
    using Nano = cuda::std::chrono::duration<double, cuda::std::nano>;
    Nano cute_t = Nano::zero();
    Nano raw_t = Nano::zero();
    auto t = cute::make_tensor(cute::make_gmem_ptr(foo), cute::make_shape(peers, cute::make_shape(cute::make_shape(stages, cells), tokens)), cute::LayoutRight{});
    if(shouldPersist){
        cuda::associate_access_property(foo, cuda::access_property::persisting{});
    }

    CUTE_UNROLL
    for(unsigned int i = 0; i < 1000; ++i){
        uint64_t start = 0, end = 0;
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        &t(0, cute::make_coord(cute::make_coord(0,1),0));
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        cute_t += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);

        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        getTokenPointer(foo, 0, 0, 1, 0);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        raw_t += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);
    }

    using BlockReduce = hipcub::BlockReduce<Nano, THREADS_PER_BLOCK>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    cute_t = BlockReduce(temp_storage).Reduce(cute_t, hipcub::Max());
    raw_t = BlockReduce(temp_storage).Reduce(raw_t, hipcub::Max());
    if(aristos::block::threadID() == 0 && !skip){
        printf("Block Id is %u, cute_t: {T: %f, V: %d, Micro: %f}, raw_t: {T: %f, V:%d, Micro: %f} "
               "persist: %s\n",
               aristos::block::threadID(),
               (cute_t / (1000*1.0)).count(),
               t(0, cute::make_coord(cute::make_coord(0,1),0)),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(cute_t/(1000*1.0)).count(),
               (raw_t / (1000*1.0)).count(),
               *getTokenPointer(foo, 0, 0, 1, 0),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(raw_t/(1000*1.0)).count(),
               (shouldPersist)? "Yes" : "No");
    }
}

__device__ unsigned int testStages = 0;
__global__ void benchBarrier(unsigned int* b, cuda::barrier<cuda::thread_scope_device>* bar, unsigned int n, bool skip = false, bool persist = false){
    using Nano = cuda::std::chrono::duration<double, cuda::std::nano>;
    Nano bar_ptr = Nano::zero();
    Nano bar_obj = Nano::zero();
    if(persist){
        cuda::associate_access_property(b, cuda::access_property::persisting{});
        cuda::associate_access_property(&testStages, cuda::access_property::persisting{});
        cuda::associate_access_property(bar, cuda::access_property::persisting{});
    }
    /*aristos::barrier::init(n, persist);*/
    constexpr auto iter = 1024;
    CUTE_UNROLL
    for(unsigned int i = 0; i < iter; ++i){
        uint64_t start = 0, end = 0;
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        if(!aristos::block::threadID()){
            auto nextStage = aristos::atomicLoad(&testStages) + 1;
            /// Arrive
            if((atomicAdd(b, 1U) + 1) == n * nextStage){
                atomicAdd(&testStages, 1U);
                /// Could execute completion function here
            }
            else{
                /// You could do some other task prior to waiting
                /// Wait
                while(aristos::atomicLoad(&testStages) != nextStage){
                    __nanosleep(2);
                }
                /// Could execute completion function here
            }
            /*aristos::barrier::wait(aristos::barrier::arrive());*/
        }
        __syncthreads();
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        bar_ptr += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);
    }

    CUTE_UNROLL
    for(unsigned int i = 0; i < iter; ++i){
        uint64_t start = 0, end = 0;
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        if(!aristos::block::threadID()){
            bar->arrive_and_wait();
        }
        __syncthreads();
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        bar_obj += static_cast<cuda::std::chrono::duration<double, cuda::std::nano>>(end - start);
    }

    if(!aristos::block::threadID() && !skip){
        printf("Block Id is %u, bar_ptr: {T: %f}, bar_obj: {T: %f}, persist: %s\n",
               aristos::grid::blockID(),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(bar_ptr/(iter*1.0)).count(),
               static_cast<cuda::std::chrono::duration<double, cuda::std::micro>>(bar_obj/(iter*1.0)).count(),
               (persist)? "Yes" : "No");
    }
}

template <class DispatchPolicy,
        class TMMA, class TiledCopyA, class TiledCopyB,
                class ElementA = cute::half_t, class ElementB = cute::half_t,
        class ElementC = cute::half_t,unsigned int stages = 1>
__global__ void processorSpec(DispatchPolicy dispatchPolicy, ElementA* A,
                              ElementA* B, ElementC* C,
                              TMMA tmma, TiledCopyA tca, TiledCopyB tcb,
                              const int& M, const int& N, const int& K){
    using namespace cute;
    using ProblemShape = decltype(make_shape(M, N, K));
    using StrideA = Underscore;
    using StrideB = Underscore;
    using StrideC = Underscore;
    using tiledMma = TMMA;
    using GmemCopyA = TiledCopyA;
    using SmemLayoutAtomA = decltype(
    composition(Swizzle<1,2,3>{},
                Layout<Shape<Int<aristos::bM>, Int<aristos::bK>>>{}));
    using SmemCopyAtomA = Copy_Atom<SM75_U32x4_LDSM_N, ElementA>;
    using TransformA = identity; // upcast from fp8 to fp16
    using GmemCopyB = TiledCopyB;
    using SmemLayoutAtomB = decltype(
    composition(Swizzle<1,2,3>{},
                Layout<Shape<Int<aristos::bN>, Int<aristos::bK>>>{}));
    using SmemCopyAtomB = Copy_Atom<SM75_U32x4_LDSM_N, ElementB>;
    using TransformB = identity; // upcast from fp8 to fp16
    using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveMma<
            DispatchPolicy,
            ProblemShape,
            ElementA,
            StrideA,
            ElementB,
            StrideB,
            tiledMma,
            GmemCopyA,
            SmemLayoutAtomA,
            SmemCopyAtomA,
            TransformA,
            GmemCopyB,
            SmemLayoutAtomB,
            SmemCopyAtomB,
            TransformB>;
    auto problemShape = ProblemShape{};
    auto ctaTiler = make_shape(aristos::bM, aristos::bN, aristos::bK);
    auto ma = make_tensor(make_gmem_ptr(A), select<0,2>(problemShape), StrideA{});
    auto mb = make_tensor(make_gmem_ptr(B), select<1,2>(problemShape), StrideB{});
    auto mc = make_tensor(make_gmem_ptr(C), select<0,1>(problemShape), StrideC{});
    auto altGridDimX = cute::ceil_div(get<0>(problemShape), cute::get<0>(ctaTiler));
    auto cta_coord = make_coord((blockIdx.x % altGridDimX), (blockIdx.x / altGridDimX), _);
    auto gA = local_tile(ma, ctaTiler, cta_coord, Step<_1, X,_1>{});
    auto gB = local_tile(mb, ctaTiler, cta_coord, Step< X,_1,_1>{});
    auto gC = local_tile(mc, ctaTiler, cta_coord, Step<_1,_1, X>{});
    auto k_tile_iter = cute::make_coord_iterator(size<2>(gA));
    int k_tile_count = size<2>(gA);
    auto accum = partition_fragment_C(tmma, Shape<Int<aristos::bM>, Int<aristos::bN>>{});
    clear(accum);
    extern __shared__ cuda::std::byte sharedBuf[];
    
    CollectiveMainloop  expert;
    expert(accum, gA, gB, accum, k_tile_iter, k_tile_count, Underscore{}, threadIdx.x, sharedBuf);
}

template<typename T>
void pop_println(std::string_view rem, T& pq)
{
    std::cout << rem << ": ";
    for (; !pq.empty(); pq.pop())
        std::cout << pq.top().toString() << ' ';
    std::cout << '\n';
}
int main() {
    /*auto size = 64;
    unsigned int* p;
    cuda::barrier<cuda::thread_scope_device>* b;
    auto host_b = new cuda::barrier<cuda::thread_scope_device>{size};
    CUTE_CHECK_ERROR(hipMallocAsync(&b, sizeof(cuda::barrier<cuda::thread_scope_device>), hipStreamPerThread));
    CUTE_CHECK_ERROR(hipMallocAsync(&p, sizeof(unsigned int), hipStreamPerThread));
    CUTE_CHECK_ERROR(hipMemcpyAsync(b, host_b, sizeof(cuda::barrier<cuda::thread_scope_device>), hipMemcpyHostToDevice, hipStreamPerThread));
    CUTE_CHECK_ERROR(hipMemsetAsync(p, 0, sizeof(unsigned int), hipStreamPerThread));
    CUTE_CHECK_LAST();

    CUTE_UNROLL
    for(int i = 0; i < 16; ++i){
        benchBarrier<<<size, THREADS_PER_BLOCK>>>(p, b, size, true);
    }
    benchBarrier<<<size, THREADS_PER_BLOCK>>>(p, b, size, false);
    benchBarrier<<<size, THREADS_PER_BLOCK>>>(p, b, size, false, true);
    CUTE_CHECK_LAST();
    CUTE_CHECK_ERROR(hipFreeAsync(p, hipStreamPerThread));
    CUTE_CHECK_ERROR(hipFreeAsync(b, hipStreamPerThread));
    free(host_b);*/

    /// Necessary to use path halving to ensure amortized "practical constant" time
    /*using djs = boost::disjoint_sets_with_storage<boost::identity_property_map,
    boost::identity_property_map, boost::find_with_path_halving>;
    auto constexpr n = 5;
    djs groups(n);
    for(int i = 0; i < n; ++i){
        groups.make_set(i);
    }
    auto p = groups.parents();
    std::cout << "Before Merges: ";
    aristos::printContainer(p);
    std::cout << std::endl;
    groups.link(0, 1);
    std::cout << "Merged 0 and 1: ";
    p = groups.parents();
    aristos::printContainer(p);
    std::cout << std::endl;
    groups.link(2, 3);
    std::cout << "Merged 2 and 3: ";
    p = groups.parents();
    aristos::printContainer(p);
    std::cout << std::endl;

    std::unordered_map<decltype(p)::value_type, std::vector<int>> sets{};
    for(int i = 0; i < n; ++i){
        sets[p[i]].push_back(i);
    }

    aristos::printMapCV(sets);
    std::cout << ']' << std::endl;
    std::cout << aristos::Streamable<decltype(sets)::key_type> << std::endl;*/

    const std::vector data = {{aristos::Edge(0,1,0.3), aristos::Edge(0,2,0.2)}};
    std::priority_queue q(data.begin(), data.end(), std::greater<>());
    pop_println("Min Priority Queue", q);
    auto b = data[1] > data[0];
    std::cout << b << std::endl;
}